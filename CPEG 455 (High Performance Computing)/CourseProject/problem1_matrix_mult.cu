#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#define BLOCK_SIZE 16

//This program can be compiled in the terminal with the command: nvcc problem1_matrix_mult.cu
//This program can be ran by running the executable a.out with the command: ./a.out

/*
    Computes dot product of two matrices in GPU

    a = GPU device pointer to a m X n matrix
    b = GPU device pointer to a n X k matrix
    c = an m X k matrix to store the result

    assigns one thread to compute one element of matrix C. Each thread loads one row of matrix A and one column of matrix B from global memory, 
	
    stores the result back to matrix C in the global memory

    the amount of computation = 2 * m * n * k flops

    the amount of global memory accesses = 2 * m * n * k
*/

__global__ void gpu_matrix_mult(int *a, int *b, int *c, int m, int n, int k){ 

    int row = blockIdx.y * blockDim.y + threadIdx.y; 

    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int sum = 0;

    if( col < k && row < m) {
        for(int i = 0; i < n; i++) {
            sum += a[row * n + i] * b[i * k + col];
        }

        c[row * k + col] = sum;
    }
} 



/*
    Tiled version of above for GPU

    only for square matrix mutiplication

    d_a = GPU pointer to a m X n matrix
    d_b = GPU pointer to a n X k matrix
    d_result = an m X k matrix to store the result

    the amount of computation is 2 x M x N x K flop. 
    
    Using tile size of B, the amount of global memory access is 2 x M x N x K / B

    
*/


__global__ void gpu_square_matrix_mult(int *d_a, int *d_b, int *d_result, int n) {

    __shared__ int tile_a[BLOCK_SIZE][BLOCK_SIZE];

    __shared__ int tile_b[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;

    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    int tmp = 0;

    int idx;

    for(int j = 0; j < gridDim.x; j++){

        idx = row * n + j * BLOCK_SIZE + threadIdx.x;

        if(idx >= n*n){
           tile_a[threadIdx.y][threadIdx.x] = 0;
        }

        else{
            tile_a[threadIdx.y][threadIdx.x] = d_a[idx];
        }

        idx = (j * BLOCK_SIZE + threadIdx.y) * n + col;
        
	if(idx >= n*n){
            tile_b[threadIdx.y][threadIdx.x] = 0;
        }  
        else{
            tile_b[threadIdx.y][threadIdx.x] = d_b[idx];
        }
        
	__syncthreads();

        for (int k = 0; k < BLOCK_SIZE; ++k){
            tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
        }
        __syncthreads();
    }

    if(row < n && col < n){
        d_result[row * n + col] = tmp;
    }
}


int main(int argc, char const *argv[]){

    int m, n, k;

    srand(3333);

    printf("What is m n and k ? (Square matricies only)\n");

    scanf("%d %d %d", &m, &n, &k);

    // allocate memory in host RAM
    int *h_a, *h_b, *h_c, *h_cc;

    hipHostMalloc((void **) &h_a, sizeof(int)*m*n);

    hipHostMalloc((void **) &h_b, sizeof(int)*n*k);

    hipHostMalloc((void **) &h_c, sizeof(int)*m*k);

    hipHostMalloc((void **) &h_cc, sizeof(int)*m*k);

    
    //making a random matrix h_a
    for (int i = 0; i < m; ++i){
        for (int j = 0; j < n; ++j) {
            h_a[i * n + j] = rand() % 1024;
        }
    }
    

    //making a random matrix h_b
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < k; ++j) {
            h_b[i * k + j] = rand() % 1024;
        }
    }

    float gpu_elapsed_time_ms, gpu_square_elapsed_time_ms;

    //Used to calculate execution time
    hipEvent_t start, stop;

    hipEventCreate(&start);

    hipEventCreate(&stop);


    // Allocate memory space on the device 
    int *d_a, *d_b, *d_c, *d_cc;
    hipMalloc((void **) &d_a, sizeof(int)*m*n);
    hipMalloc((void **) &d_b, sizeof(int)*n*k);
    hipMalloc((void **) &d_c, sizeof(int)*m*k);
    hipMalloc((void **) &d_cc, sizeof(int)*m*k);

    // copy matrix A and B from host to device memory
    hipMemcpy(d_a, h_a, sizeof(int)*m*n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(int)*n*k, hipMemcpyHostToDevice);

    unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
   
    

    //Can only run if the matrices are square because we need to compare the regular time to the square time
    if(m == n && n == k){

        // start to count execution time of tiled GPU version
        hipEventRecord(start, 0);

        //running it 100 times to get an average
        for(int i = 0; i <100; i++){
            gpu_square_matrix_mult<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, n);
        }
            
        //Stop recording time
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        // compute time elapse on GPU function
        hipEventElapsedTime(&gpu_square_elapsed_time_ms, start, stop);


        // start to count execution time of regular GPU version
        hipEventRecord(start, 0);

        //Non-tiled version, again run it 100 times
        for(int i = 0; i <100; i++){
            gpu_matrix_mult<<<dimGrid, dimBlock>>>(d_a, d_b, d_cc, m, n, k);
        }    

        //Stop recording time
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        // compute time elapse on standard GPU function
        hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
         

        // Transfer results from device to host 
        hipMemcpy(h_c, d_c, sizeof(int)*m*k, hipMemcpyDeviceToHost);
        hipMemcpy(h_cc, d_cc, sizeof(int)*m*k, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
    }

    else{
        printf("Error: Please use a square matrix\n");
        exit(0);
    }

    //divide by 100 because we ran 100 times
    printf("Average time elapsed on standard matrix multiplication of %dx%d . %dx%d on GPU: %f ms.\n\n", m, n, n, k, gpu_elapsed_time_ms/100);
    //divide by 100 because we ran 100 times
    printf("Average time elapsed on tiled matrix multiplication of %dx%d . %dx%d on GPU: %f ms.\n\n", m, n, n, k, gpu_square_elapsed_time_ms/100);

    printf("Average speedup Percentage from nontiled to tiled algorithim = %f%s\n", 100*gpu_elapsed_time_ms / gpu_square_elapsed_time_ms, "%");


    // validate results
    int all_ok = 1;
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < k; ++j)
        {      
            if(h_cc[i*k + j] != h_c[i*k + j])
            {
                all_ok = 0;
            }
            
        }   
    }
    printf("\n");

    if(all_ok)
    {
        printf("all results are correct!!!\n");
    }
    else
    {
        printf("incorrect results\n");
    }



    // free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_cc);
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    hipHostFree(h_cc);
    return 0;
}
